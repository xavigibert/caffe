#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

#define EPSILON (1e-6)

namespace caffe {

template <typename Dtype>
__global__ void kernel_norm(int m, int k, const Dtype* D, Dtype* diagDtD) {
  CUDA_KERNEL_LOOP(index, k) {
    Dtype res = (Dtype)0.;
    for (int j =0; j < m; ++j)
      res += D[index+j*k] * D[index+j*k];
    diagDtD[index] = res;
  }
}

template <typename Dtype>
void DictionaryLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  Forward_cpu(bottom, top);
//  // Perform normalization and decomposition (if necessary)
//  forward_preprocess_cpu();
//  // Perform sparse coding (and optionally dictionary learning) on each input vector
//  const Dtype* D = this->blobs_[0]->gpu_data();
//  const Dtype* Vt = this->blobs_[bias_idx_ + 2]->gpu_data();
//  const Dtype* Vt_s2 = this->blobs_[bias_idx_ + 3]->gpu_data();
//  for (int i = 0; i < top.size()/2; ++i) {
//    const Dtype* bottom_data = bottom[i]->gpu_data();
//    Dtype* top_data = top[2*i]->mutable_gpu_data();
//    Dtype* loss = top[2*i+1]->mutable_gpu_data();
//    caffe_gpu_set(1, (Dtype)0., loss);
//    for (int n = 0; n < this->num_; ++n) {
//      // Perform forward sparse coding
//      this->forward_gpu_sparse_coding(bottom_data + bottom[i]->offset(n),
//          D, Vt, Vt_s2, top_data + top[2*i]->offset(n), loss);
//      if (this->bias_term_) {
//        const Dtype* bias = this->blobs_[bias_idx_]->gpu_data();
//        this->forward_gpu_bias(top_data + top[i]->offset(n), bias);
//      }
//    }
//    // Scale objective value in second output
//    caffe_gpu_scal(1, (Dtype)1./(Dtype)(num_*conv_out_spatial_dim_), loss);
//  }
}

// A is mxk, B is kxm
template <typename Dtype>
__global__ void transpose_kernel(int n, int m, int k, const Dtype* A, Dtype* B) {
  CUDA_KERNEL_LOOP(index, n) {
    int idx_row = index / m;
    int idx_col = index % m;
    B[index] = A[idx_row + idx_col*k];
  }
}

// Perform B = A^T
template<typename Dtype>
void DictionaryLayer<Dtype>::transpose_gpu(int m, int k, const Dtype* A, Dtype* B) {
  int N = m*k;
  transpose_kernel<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, m, k, A, B);
}

template <typename Dtype>
__global__ void kernel_compute_diag_weights(int n, Dtype lambda,
      const Dtype* vec_alpha, Dtype* diag) {
  CUDA_KERNEL_LOOP(index, n) {
    diag[index] = 2*lambda/(fabs(vec_alpha[index])+EPSILON);
  }
}

template <typename Dtype>
__global__ void kernel_hard_threshold(int n, Dtype lambda, Dtype* vec_alpha) {
  CUDA_KERNEL_LOOP(index, n) {
    if (fabs(vec_alpha[index]) < lambda)
      vec_alpha[index] = (Dtype)0.;
  }
}

// Perform sparse coding on the GPU, estimate the loss and add it to the
// previous loss value
template <typename Dtype>
void DictionaryLayer<Dtype>::forward_gpu_sparse_coding(const Dtype* input,
      const Dtype* D, const Dtype* Vt, const Dtype *Vt_s2, Dtype* output,
      Dtype* loss, bool skip_im2col) {
  const Dtype* col_buff = input;
  if (!is_1x1_) {
    if (!skip_im2col) {
      conv_im2col_gpu(input, col_buffer_.mutable_gpu_data());
    }
    col_buff = col_buffer_.gpu_data();
  }
  // Perform sparse coding for each input vector
  int m = kernel_dim_;
  int k = num_output_;
  Dtype* vec_d = vec_d_buffer_.mutable_gpu_data();      // D^T * x
  Dtype* vec_r = vec_r_buffer_.mutable_gpu_data();      // Residual vector
  Dtype* vec_p = vec_p_buffer_.mutable_gpu_data();      // Descent direction
  Dtype* vec_w = vec_w_buffer_.mutable_gpu_data();      // Vector w
  Dtype* sparse_codes = sparse_codes_buffer_.mutable_gpu_data();
  Dtype* diag = tmp_buffer_.mutable_gpu_data() + std::max(k,m);
  Dtype* tmp = tmp_buffer_.mutable_gpu_data() + 2*std::max(k,m);
  for (int i = 0; i < conv_out_spatial_dim_; ++i)
  {
    const Dtype* x = col_buff + i*m;  // Input sample
    Dtype* vec_alpha = sparse_codes + i*k;   // Sparse code vector
    // Initialize sparse code
    caffe_gpu_set<Dtype>(k, (Dtype)1., vec_alpha);
    // Perform num_iter_irls iterations of iteratively reweighted
    // least squares using the previous result as starting value
    for (int iter_irls = 0; iter_irls < num_iter_irls_; ++iter_irls)
    {
      // Build matrix w = diag(2*lambda/fabs(alpha[])
      kernel_compute_diag_weights<Dtype><<<CAFFE_GET_BLOCKS(k),
          CAFFE_CUDA_NUM_THREADS>>>(k, (Dtype)lambda_, vec_alpha, diag);
      // Build vector d = D^T * x
      caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, k, 1, m,
           (Dtype)1., D, x,
           (Dtype)0., vec_d);
      // Perform conjugate gradient descent to approximately solve
      // C * alpha = d     for alpha
      // Note: We do not compute matrix C explicitly, since
      //   C * alpha = diag(2*lambda/fabs(alpha[]) .* alpha + V * Vt_s2 * alpha
      //   C * alpha = tmp + V * Vt_s2 * alpha
      // is more efficient to compute
      conjugate_gradient_gpu(k, rank_, diag, Vt, Vt_s2, vec_d, vec_alpha,
          num_iter_cg_, vec_p, vec_r, vec_w, tmp);
    }
    // Apply hard threshold to sparse codes
    kernel_hard_threshold<<<CAFFE_GET_BLOCKS(k), CAFFE_CUDA_NUM_THREADS>>>(
        k, (Dtype)lambda_, vec_alpha);
    //add_objective_gpu(m, k, D, x, vec_alpha, loss);
  }
  // Sparse codes are in pixel-first order, we need to transpose them so they
  // are in channel-first order
  transpose_gpu(conv_out_spatial_dim_, num_output_, sparse_codes, output);
}

template <typename Dtype>
void DictionaryLayer<Dtype>::forward_gpu_bias(Dtype* output,
    const Dtype* bias) {
  int k = num_output_;
  caffe_gpu_add(k, bias, output, output);
}

template <typename Dtype>
void DictionaryLayer<Dtype>::add_objective_gpu(int m, int k,
        const Dtype* D, const Dtype* x, const Dtype* alpha,
        Dtype* loss) {
  // Compute objective function
  // Cost(alpha) = 0.5*||x_t-D*alpha||_2^2 + lambda*||alpha||_1
  Dtype* tmp = tmp_buffer_.mutable_gpu_data();
  caffe_gpu_memcpy(m, x, tmp);
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, m, 1, k,
                        (Dtype)(-1.), D, alpha,
                        (Dtype)1., tmp);
  Dtype cost = (Dtype)0.;
  //caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, 1, 1, m, (Dtype)1., tmp,
  //    tmp, (Dtype)0., cost);
  caffe_gpu_dot<Dtype>(m, tmp, tmp, &cost);
  //caffe_gpu_scale<Dtype>(1, 0.5, cost, cost);
  Dtype asum = (Dtype)0.;
  // Sum of absolute values of elements in alpha
  caffe_gpu_asum<Dtype>(k, alpha, &asum);
  //caffe_gpu_scale<Dtype>(1, Dtype(lambda_), asum, asum);
  // Add cost and asum to loss
  //caffe_gpu_add<Dtype>(1, asum, loss, loss);
  //caffe_gpu_add<Dtype>(1, cost, loss, loss);
  cost = 0.5 * cost + lambda_ * asum;
  //LOG(INFO) << "COST = " << cost;
  caffe_gpu_add_scalar(1, cost, loss);
}

// Compute C * x = w .* x + V * Vt2 * x
//         (kx1)    (kx1)  (k*r)(r*k)(k*1)
template <typename Dtype>
void DictionaryLayer<Dtype>::compute_Cx_gpu(int k, int r, const Dtype* w,
      const Dtype* Vt, const Dtype* Vt2, const Dtype* x,
      Dtype* tmp, Dtype* Cx) {
  caffe_gpu_mul(k, w, x, Cx);
  caffe_gpu_gemv<Dtype>(CblasNoTrans, r, k, (Dtype)1., Vt2, x, (Dtype)0.,
      tmp);
  caffe_gpu_gemv<Dtype>(CblasTrans, r, k, (Dtype)1., Vt, tmp, (Dtype)1., Cx);
}

template <typename Dtype>
void DictionaryLayer<Dtype>::conjugate_gradient_gpu(int k, int r,
      const Dtype* weights, const Dtype* Vt, const Dtype* Vt2, const Dtype* d,
      Dtype* x, int num_iter, Dtype* temp_p, Dtype* temp_r, Dtype* temp_w,
      Dtype* tmp) {
  // Initialize the residual
  compute_Cx_gpu(k, r, weights, Vt, Vt2, x, tmp, temp_r);
  caffe_gpu_sub(k, d, temp_r, temp_r);
  // Compute norm of the residual
  Dtype prev_norm_r = (Dtype)0.;
  caffe_gpu_dot<Dtype>(k, temp_r, temp_r, &prev_norm_r);
  if (fabs(prev_norm_r) < EPSILON) {
    return;   // Accept initial solution
  }
  // Initialize the descent direction
  caffe_gpu_memcpy(k*sizeof(Dtype), temp_r, temp_p);
  // Perform num_iter_cg iterations of conjugate gradient descent
  for (int iter_cg = 0; iter_cg < num_iter; ++iter_cg)
  {
    // w = C * p
    compute_Cx_gpu(k, r, weights, Vt, Vt2, temp_p, tmp, temp_w);
    Dtype dot_p_w = (Dtype)0.;
    caffe_gpu_dot<Dtype>(k, temp_p, temp_w, &dot_p_w);
    Dtype alpha = prev_norm_r / dot_p_w;
    CHECK(!isnan(alpha));
    // x = x + alpha*p
    caffe_gpu_axpy<Dtype>(k, alpha, temp_p, x);
    // r = r - alpha*w
    caffe_gpu_axpy<Dtype>(k, -alpha, temp_w, temp_r);
    // Compute norm of new residual
    Dtype norm_r = (Dtype)0.;
    caffe_gpu_dot<Dtype>(k, temp_r, temp_r, &norm_r);
    // Compute beta
    Dtype beta = norm_r / prev_norm_r;
    CHECK(!isnan(beta));
    // p = r + beta*p
    caffe_gpu_axpby<Dtype>(k, (Dtype)1., temp_r, beta, temp_p);
    prev_norm_r = norm_r;
    if (fabs(prev_norm_r) < EPSILON) {
      return;
    }
  }
}

template <typename Dtype>
__global__ void kernel_mod_gradient(int k, const Dtype* alpha,
    const Dtype* alpha_diff, Dtype* mod_alpha_diff) {
  CUDA_KERNEL_LOOP(index, k) {
    mod_alpha_diff[index] = alpha[index] == (Dtype)0.
        ? (Dtype)0. : alpha_diff[index];
  }
}

template <typename Dtype>
void DictionaryLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
//  Backward_cpu(top, propagate_down, bottom);
  const Dtype* D = this->blobs_[0]->gpu_data();
  Dtype* D_diff = this->blobs_[0]->mutable_gpu_diff();
  if (this->param_propagate_down_[0]) {
    caffe_gpu_set(this->blobs_[0]->count(), Dtype(0), D_diff);
  }
  CHECK(is_dict_normalized_);
  CHECK_EQ(conv_out_spatial_dim_, 1) << "Convolutional dictionaries not implemented, yet!";
  // Temporary storage and precomputed constants
  int m = kernel_dim_;
  int k = num_output_;
  Dtype* tmp1 = tmp_buffer_.mutable_gpu_data();
  Dtype* tmp2 = tmp_buffer_.mutable_gpu_data() + std::max(k,m);
  Dtype* tmp3 = tmp_buffer_.mutable_gpu_data() + 2*std::max(k,m);
  Dtype* tmp_dl_dx = tmp_buffer_.mutable_gpu_data() + 3*std::max(k,m);
  // Precomputed matrices
  const Dtype* Vt = this->blobs_[bias_idx_ + 2]->gpu_data();
  const Dtype* Vt_sn2 = Vt_sn2_buffer_.gpu_data();
  const Dtype* Ddagger = Ddagger_buffer_.gpu_data();
  for (int idx = 0; idx < top.size()/2; ++idx) {
    const Dtype* top_diff = top[2*idx]->gpu_diff();
    const Dtype* top_data = top[2*idx]->gpu_data();
    const Dtype* bottom_data = bottom[idx]->gpu_data();
    Dtype* bottom_diff = bottom[idx]->mutable_gpu_diff();
    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      Dtype* bias_diff = this->blobs_[bias_idx_]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        this->backward_gpu_bias(bias_diff, top_diff + top[idx*2]->offset(n));
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[idx]) {
      for (int n = 0; n < this->num_; ++n) {
        const Dtype* alpha = top_data + top[idx*2]->offset(n);
        const Dtype* alpha_diff = top_diff + top[idx*2]->offset(n);
        // Precompute modified output gradient
        Dtype* mod_alpha_diff = mod_alpha_diff_buffer_.mutable_gpu_data();
        kernel_mod_gradient<Dtype><<<CAFFE_GET_BLOCKS(k),
            CAFFE_CUDA_NUM_THREADS>>>(k, alpha, alpha_diff, mod_alpha_diff);
        // dl/dx is necessary for both gradients
        Dtype* dl_dx = propagate_down[idx] ?
              bottom_diff + bottom[idx]->offset(n) : tmp_dl_dx;
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[idx] || this->param_propagate_down_[0]) {
          this->backward_gpu_gemm(mod_alpha_diff, Ddagger, dl_dx);
        }
        // gradient w.r.t. dictionary. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
          this->dict_gpu_backprop(bottom_data + bottom[idx]->offset(n),
              dl_dx, mod_alpha_diff, Ddagger, Vt, Vt_sn2, tmp1, tmp2, tmp3,
              D_diff);
          this->dict_gpu_optimize(bottom_data + bottom[idx]->offset(n), alpha,
              D, (Dtype)etha_, tmp1, tmp2, D_diff);
          // Mark dictionary as unnormalized
          is_dict_normalized_ = false;
          Dtype* Dflag = this->blobs_[bias_idx_ + 1]->mutable_cpu_data();
          *Dflag = (Dtype)1.;
        }
      }
    }
  }
}

template <typename Dtype>
void DictionaryLayer<Dtype>::dict_gpu_backprop(const Dtype* x, const Dtype* dl_dx,
      const Dtype* mod_alpha_diff, const Dtype* Dtdagger, const Dtype* Vt,
      const Dtype* Vt_sn2, Dtype* tmp1, Dtype* tmp2, Dtype* tmp3, Dtype* D_diff) {
  int m = kernel_dim_;
  int k = num_output_;
  int r = rank_;
  // Compute intermediate products
  // tmp1 = x^T * (D^dagger)^T
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, 1, k, m, (Dtype)1., x,
      Dtdagger, (Dtype)0., tmp1);
  // tmp2 = dl_dalpha * V
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, 1, r, k, (Dtype)1.,
      mod_alpha_diff, Vt, (Dtype)0., tmp2);
  // tmp3 = tmp2 * Vt_sn2
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, 1, k, r, (Dtype)1.,
      tmp2, Vt_sn2, (Dtype)0., tmp3);
  // Compute gradient of dictionary and add it to D_diff
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, m, k, 1, -(Dtype)2., dl_dx,
      tmp1, (Dtype)1., D_diff);
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, m, k, 1, (Dtype)1., x,
      tmp3, (Dtype)1., D_diff);
  // Result to be D_diff = D_diff - 2 * (D^dagger)^T * x^T * mod_alpha_diff^t * (D^dagger)^T
  //                              + x * mod_alpha_diff * V * Vt_sn2

}

template <typename Dtype>
void DictionaryLayer<Dtype>::dict_gpu_optimize(const Dtype* x,
    const Dtype* alpha, const Dtype* D, Dtype etha, Dtype* tmp1, Dtype* tmp2,
    Dtype* D_diff) {
  if (etha == (Dtype)0.)
    return;
  int m = kernel_dim_;
  int k = num_output_;
  caffe_copy(m, x, tmp1);
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, m, 1, k,
      (Dtype)1., D, alpha, -(Dtype)1., tmp1);
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, m, k, 1,
      etha, tmp1, alpha, (Dtype)1., D_diff);
}

template <typename Dtype>
void DictionaryLayer<Dtype>::backward_gpu_gemm(const Dtype* mod_alpha_diff,
    const Dtype* D, Dtype* input) {
  Dtype* col_buff = col_buffer_.mutable_gpu_data();
  if (is_1x1_) {
    col_buff = input;
  }
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, kernel_dim_,
      1, num_output_,
      (Dtype)1., D, mod_alpha_diff,
      (Dtype)0., col_buff);
  if (!is_1x1_) {
    conv_col2im_gpu(col_buff, input);
  }
}

template <typename Dtype>
void DictionaryLayer<Dtype>::backward_gpu_bias(Dtype* bias,
    const Dtype* input) {
  int k = num_output_;
  caffe_gpu_add(k, input, bias, bias);
}

INSTANTIATE_LAYER_GPU_FUNCS(DictionaryLayer);

}  // namespace caffe
